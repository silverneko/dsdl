
#include <hip/hip_runtime.h>
#include <algorithm>
#include <iostream>
#include <cassert>
#include <cmath>
#include <chrono>

using namespace std;

#define TIMER_SET(t0) std::chrono::time_point<std::chrono::steady_clock> t0 = std::chrono::steady_clock::now()
#define TIMER_DIFF(t0, t1) std::chrono::duration_cast<std::chrono::microseconds> (t1 - t0).count()

__global__
void kernel(float * A, int N) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id >= N) return;
  A[id] = sqrtf( powf(3.1415926f, id) );
}

int main(int argc, char * argv[]) {
  const int N = 10000000;
  float * A;

  TIMER_SET(t0);

#ifdef CPU_DEMO
  A = new float[N];
#else
  hipMallocManaged(&A, sizeof(float) * N, hipMemAttachGlobal);
#endif

  // Calculate A[i] = sqrt( pow(PI, i) ) (add two vectors)
#ifdef CPU_DEMO
  for (int i = 0; i < N; ++i) {
    A[i] = sqrtf( powf(3.1415926f, i) );
  }
#else
  int dimBlock = 64;
  int dimGrid = (N + dimBlock - 1) / dimBlock;
  kernel<<<dimGrid, dimBlock>>>(A, N);
  hipDeviceSynchronize();
#endif

  // for (int i = 0; i < 5; ++i) {
  //   cout << ' ' << A[i];
  // }
  // cout << endl;

  TIMER_SET(t1);
  cout << "Time (microsecond): " << TIMER_DIFF(t0, t1) << endl;

#ifdef CPU_DEMO
  delete A;
#else
  hipFree(A);
#endif

  return 0;
}
