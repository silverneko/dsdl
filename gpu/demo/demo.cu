
#include <hip/hip_runtime.h>
#include <algorithm>
#include <iostream>
#include <cassert>
#include <cmath>

using namespace std;

__global__
void kernel1(int * A, int * B, int N) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id >= N) return;
  A[id] += B[id];
}

int main(int argc, char * argv[]) {
  const int N = 10000;
  int * A, * B;

  // A = new int[N];
  // B = new int[N];
  hipMallocManaged(&A, sizeof(int) * N, hipMemAttachGlobal);
  hipMallocManaged(&B, sizeof(int) * N, hipMemAttachGlobal);

  // Do something with A and B ...

  // Calculate A = A + B (add two vectors)
  int dimBlock = 32;
  int dimGrid = N / 32 + 1;
  kernel1<<<dimGrid, dimBlock>>>(A, B, N);
  hipDeviceSynchronize();

  // for (int i = 0; i < N; ++i) {
  //   A[i] += B[i];
  // }

  hipFree(A);
  hipFree(B);
  return 0;
}
